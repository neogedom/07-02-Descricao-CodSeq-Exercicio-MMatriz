#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void helloFromGPU (int n)
{
	printf("Hello from GPU with grid %d, block %d, and thread %d\n", n, blockIdx.x, threadIdx.x);
	//printf("From:%d, %d ", n, blockIdx.x);
}

int main (void)
{
	helloFromGPU<<<1,10>>>(1);

	hipDeviceSynchronize();

	helloFromGPU<<<10,1>>>(2);

	hipDeviceSynchronize();

	printf("Hello CPU\n");

	return 0;
}
