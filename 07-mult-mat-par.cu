#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#define BLOCK_SIZE 16

__global__ void mult (double *a, double *b, double *c) {
    int tam = blockDim.x;
    int row=blockIdx.x*tam+threadIdx.x;
    int col=blockIdx.x*tam+threadIdx.x;

    if (row < tam && col < tam) {
        int sum = 0;
        for(int k=0;k<tam;k++) {
            sum += a[row*tam+k]*b[k*tam+col];
        }
        c[row*tam+col] = sum;
            
    }
}

int main(int argc,char **argv){
    //Declara as matrizes
    double *matrizA, *matrizB, *matrizC;
    double *dev_a, *dev_b, *dev_c;
    
    //Declara as variáveis de tamanho e índice
    int tam,i,j;
    printf("Teste");
    //Lê a dimensão da matriz
    fscanf(stdin,"%d\n",&tam); 
   


    //Aloca as matrizes A e B no hosto
    hipHostMalloc((void**) &matrizA, (tam*tam*sizeof(double)), hipHostMallocDefault);
    hipHostMalloc((void**) &matrizB, (tam*tam*sizeof(double)), hipHostMallocDefault);
    hipHostMalloc((void**) &matrizC, (tam*tam*sizeof(double)), hipHostMallocDefault);



    //Lê as matrizes A e B
    for(i=0;i<tam;i++)
        for(j=0;j<tam;j++)
            fscanf(stdin, "%lf ", &dev_a[i * tam + j]);
    for(i=0;i<tam;i++)
        for(j=0;j<tam;j++)
            fscanf(stdin,"%lf ",&dev_b[i*tam+j]);


    //Aloca memória na GPU
    hipMalloc( (void**)&dev_a, tam * sizeof(int));
    hipMalloc( (void**)&dev_b, tam * sizeof(int));
    hipMalloc( (void**)&dev_c, tam * sizeof(int));

    // Copia as matrizes para a memória do device
    hipMemcpy(dev_a, matrizA, sizeof(int)*tam*tam, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, matrizB, sizeof(int)*tam*tam, hipMemcpyHostToDevice);


    unsigned int grid_rows = (tam + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (tam + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    mult<<<dimGrid,dimBlock>>>(dev_a, dev_b, dev_c);


    matrizC=(double*)malloc(tam*tam*sizeof(double));
    hipMemcpy(dev_c,matrizC, sizeof(int),hipMemcpyDeviceToHost);

    //Imprime o resultado    
    for(i=0;i<tam;i++){
        for(j=0;j<tam;j++)
            printf("%.1lf ",matrizC[i*tam+j]);
        printf("\n");
    }
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    //Desaloca as matrizes
    free(matrizC);

    return 0;
}
